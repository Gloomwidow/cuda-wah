#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "bit_functions.cuh"

#define EMPTY_WORD	0
#define FULL_WORD	1
#define TAIL_WORD	2

#define WARPS_IN_BLOCK 2

namespace cg = cooperative_groups;

//__global__ void scan(float *g_odata, float *g_idata, int n)
//{
//	extern __shared__ float temp[]; // allocated on invocation
//	int thid = threadIdx.x;
//	int pout = 0, pin = 1;   
//	
//	// Load input into shared memory.
//	// This is exclusive scan, so shift right by one
//	// and set first element to 0
//	temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;
//	__syncthreads();
//	for (int offset = 1; offset < n; offset *= 2)
//	{     
//		pout = 1 - pout; // swap double buffer indices     
//		pin = 1 - pout;
//		if (thid >= offset)
//			temp[pout*n+thid] += temp[pin*n+thid - offset];
//		else
//			temp[pout*n+thid] = temp[pin*n+thid];
//		__syncthreads();
//	}   
//	g_odata[thid] = temp[pout*n+thid]; // write output
//} 

typedef struct segment {
	uchar1 l_end_type;
	uchar1 l_end_len;

	uchar1 r_end_type;
	uchar1 r_end_len;
} segment;

__global__ void SharedMemKernel(UINT* input, int inputSize, UINT* output)
{
	const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	const int lane_id = threadIdx.x % warpSize;
	const int warp_id = threadIdx.x / warpSize;
	int warps_count;// = (inputSize % warpSize == 0) ? (inputSize / warpSize) : (inputSize / warpSize) + 1;	// TODO: correct for when there are many blocks
	if ((blockIdx.x + 1)*blockDim.x > inputSize)	// last block can enter here
	{
		warps_count = (inputSize - blockIdx.x*blockDim.x) / warpSize;
		if (inputSize % warpSize != 0)
			warps_count++;
	}
	else
		warps_count = blockDim.x / warpSize;

	UINT gulp = input[thread_id];

	// calculate type of the word
	bool is_zero = is_zeros(gulp);
	bool is_one = is_ones(gulp);
	char w_type;
	if (is_zero)
		w_type = EMPTY_WORD;
	else if (is_one)
		w_type = FULL_WORD;
	else
		w_type = TAIL_WORD;

	// is this thread the beginning of a section?
	bool is_begin = false;
	char prev_type = __shfl_up_sync(FULL_MASK, w_type, 1);
	if (thread_id < inputSize)
	{
		is_begin = (w_type == TAIL_WORD) || (w_type != prev_type);
		if (lane_id == 0)
			is_begin = true;
	}

	unsigned warp_begins_mask = __ballot_sync(FULL_MASK, is_begin);

	__shared__ segment segments[WARPS_IN_BLOCK];	// TODO: make it allocated dynamically
	int segment_len;
	if (is_begin)
	{																									// find ID of the next thread-beginning and thus the length of the section
		segment_len = (lane_id == warpSize - 1) ? 0 : __ffs(warp_begins_mask >> (lane_id + 1));			// note: bit shift "(int) >> 32" is not defined
																										// note: __ffs(0) = 0
		if (segment_len == 0)	// the last thread-beginning in warp
		{
			segment_len = (warp_id < warps_count - 1) ? (warpSize - lane_id) : (warps_count*warpSize - thread_id);	
																										// considers case of the last thread-beginning in the last warp in block
																										// when inputSize is not divisible by 32
			segments[warp_id].r_end_type = make_uchar1(w_type);
			segments[warp_id].r_end_len = make_uchar1(segment_len);
		}
		if (lane_id == 0)		// the first thread-beginning in warp
		{
			segments[warp_id].l_end_type = make_uchar1(w_type);
			segments[warp_id].l_end_len = make_uchar1(segment_len);
		}
	}
	__syncthreads();

	if (is_begin)
	{
		if (warp_id > 0 && lane_id == 0 && (segments[warp_id - 1].r_end_type.x == w_type))				// check if the first thread-beginning in warp is really
			is_begin = false;																			// thread-beginning in the context of the block...

		if (segment_len == 0)																			// ...if not, the last thread-beginning form prev. warp should add sth to its `segment_len`
		{
			for (int i = warp_id + 1; i < warps_count && segments[i].l_end_type.x == w_type; i++)
			{
				segment_len += segments[i].l_end_len.x;		// check types
				if (segments[i].l_end_len.x != warpSize)
					break;
			}
		}
	}
	__syncthreads();

	// here every thread-beginning knows its segment's length (in-block boundaries)

	// in-warp scan, taken from https://github.com/NVIDIA/cuda-samples/blob/master/Samples/shfl_scan/shfl_scan.cu
	// not work-efficient implementation
	// TODO: do better implementation
	// TODO: scan should be exclusive
	int value = is_begin ? 1 : 0;
	for (int i = 1; i <= warpSize; i *= 2)
	{
		int n = __shfl_up_sync(FULL_MASK, value, i);

		if (lane_id >= i)
			value += n;
	}

	// inter-warp scan
	__shared__ int sums[WARPS_IN_BLOCK];
	if (lane_id == warpSize - 1)
		sums[warp_id] = value;
	__syncthreads();

	// the same shfl scan operation, but performed on warp sums
	// this can be safely done by a single warp
	if (warp_id == 0 && lane_id < warps_count)
	{
		int warp_sum = sums[lane_id];

		int mask = (1 << warps_count) - 1;
		for (int i = 1; i <= warps_count; i *= 2)
		{
			int n = __shfl_up_sync(mask, warp_sum, i, warps_count);
			if (lane_id >= i)
				warp_sum += n;
		}
		sums[lane_id] = warp_sum;
	}
	__syncthreads();

	if (warp_id > 0 && is_begin)
		value += sums[warp_id - 1];
	// now value is correct in block boundaries


	
	//segment* block_segments = (segment*)output;															// this allocation is being reused
	//// find the last thread-beginning in block
	//warp_begins_mask = __ballot_sync(FULL_MASK, is_begin);
	//__shared__ unsigned block_begins_masks[WARPS_IN_BLOCK];
	//if (lane_id == 0)
	//	block_begins_masks[warp_id] = warp_begins_mask;
	//__syncthreads();

	//if (warp_id == 0 && lane_id < warps_count)															// find last warp in block which contains any thread-beginning
	//{
	//	unsigned begins_mask = block_begins_masks[lane_id];
	//	unsigned is_mask_nonzero = __ballot_sync(__activemask(), begins_mask != EMPTY_MASK);
	//	block_begins_masks[0] = warpSize - 1 - __clz(is_mask_nonzero);									// write its warp_id in shared memory
	//}
	//__syncthreads();

	//bool am_last_beginning = false;
	//if (warp_id == block_begins_masks[0])		// find last thread-beginning in block
	//{
	//	int lane = warpSize - 1 - __clz(warp_begins_mask);	// lane_id of the this thread
	//	if (lane_id == lane)
	//	{
	//		am_last_beginning = true;

	//		block_segments[blockIdx.x].r_end_type = make_uchar1(w_type);
	//		block_segments[blockIdx.x].r_end_len = make_uchar1(segment_len);
	//	}
	//}
	//if (threadIdx.x == 0)						// first thread-beginning in block
	//{
	//	block_segments[blockIdx.x].l_end_type = make_uchar1(w_type);
	//	block_segments[blockIdx.x].l_end_len = make_uchar1(segment_len);
	//}
	//cg::grid_group grid = cg::this_grid();
	//grid.sync();
	////cg::sync(cg::this_grid());

	//if (blockIdx.x > 0 && threadIdx.x == 0 && block_segments[blockIdx.x - 1].r_end_type.x == w_type)	// check if the first thread-beginning in block is really
	//	is_begin = false;																				// thread-beginning in the context of the grid...

	//if (am_last_beginning)																				// ...if not, the last thread-beginning form prev. block should add sth to its `segment_len`
	//{
	//	for (int i = blockIdx.x + 1; i < gridDim.x && block_segments[i].l_end_type.x == w_type; i++)
	//	{
	//		segment_len += block_segments[i].l_end_len.x;		// check types
	//		if (segments[i].l_end_len.x != blockDim.x)
	//			break;
	//	}
	//}

	//// INTER-BLOCK SCAN
	////__shared__ int sums[WARPS_IN_BLOCK];
	//UINT* block_sums = output;
	////if (threadIdx.x == blockDim.x - 1)
	//if (am_last_beginning)
	//{
	//	printf("block %d has value: %d\n", blockIdx.x, value);
	//	block_sums[blockIdx.x] = value;
	//}
	//grid.sync();

	//thrust::inclusive_scan(thrust::device, block_sums, block_sums + gridDim.x, block_sums);
	//__syncthreads();

	// the same shfl scan operation, but performed on block sums
	/// this can be safely done by a single warp
	//if (warp_id == 0 && lane_id < warps_count)
	//{
	//	int warp_sum = sums[lane_id];

	//	int mask = (1 << warps_count) - 1;
	//	for (int i = 1; i <= warps_count; i *= 2)
	//	{
	//		int n = __shfl_up_sync(mask, warp_sum, i, warps_count);
	//		if (lane_id >= i)
	//			warp_sum += n;
	//	}
	//	sums[lane_id] = warp_sum;
	//}
	//__syncthreads();

	if (is_begin)
	{
		// gather
		//if (warp_id > 0)
		//	value += sums[warp_id - 1];

		if (w_type == EMPTY_WORD)
			output[value - 1] = get_compressed(segment_len, 0);
		else if (w_type == FULL_WORD)
			output[value - 1] = get_compressed(segment_len, 1);
		else
			output[value - 1] = gulp;
	}
}

void printBits(size_t const size, void const * const ptr)
{
	unsigned char *b = (unsigned char*)ptr;
	unsigned char byte;
	size_t i, j;

	for (i = size - 1; i >= 0; i--) {
		for (j = 7; j >= 0; j--) {
			byte = (b[i] >> j) & 1;
			printf("%u", byte);
		}
	}
	puts("");
}

void ensure_cooperativity_support()
{
	hipDeviceProp_t deviceProp = { 0 };

	int device;
	CUDA_CHECK(hipGetDevice(&device), Finish);

	CUDA_CHECK(hipGetDeviceProperties(&deviceProp, device), Finish);
	if (!deviceProp.cooperativeLaunch)
	{
		printf("\nSelected GPU (%d) does not support Cooperative Kernel Launch, Waiving the run\n", device);
		exit(EXIT_FAILURE);
	}
Finish:
}

UINT* SharedMemWAH(int size, UINT* input)//, size_t size)
{
	ensure_cooperativity_support();

	UINT* result = nullptr;

	UINT* d_input;
	UINT* d_output;
	CUDA_CHECK(hipMalloc((void**)&d_input, size * sizeof(UINT)), FreeInput);		// reinterpret_cast<>
	CUDA_CHECK(hipMalloc((void**)&d_output, size * sizeof(UINT)), Free);
	CUDA_CHECK(hipMemcpy(d_input, input, size * sizeof(UINT), hipMemcpyHostToDevice), Free);

	int threads_per_block = 64;
	int blocks = size / threads_per_block;
	if (size % threads_per_block != 0)
		blocks++;

	SharedMemKernel<<<blocks, threads_per_block>>>(d_input, size, d_output);

	CUDA_CHECK(hipGetLastError(), Free);
	CUDA_CHECK(hipDeviceSynchronize(), Free);

	UINT* output = new UINT[size];
	CUDA_CHECK(hipMemcpy(output, d_output, size * sizeof(UINT), hipMemcpyDeviceToHost), Free);
	result = output;

	printf("Sequence after global-compression:\n");
	for (int i = 0; i < size; i++)
	{
		UINT c = compressed_count(output[i]);
		if (get_bit(output[i], 0)) printf("(%u,%u) ", c, get_bit(output[i], 1));
		else printf("x ");
	}
	printf("\n");

Free:
	CUDA_CHECK(hipFree(d_output), FreeInput);
FreeInput:
	CUDA_CHECK(hipFree(d_input), Fin);
Fin:
	return result;
}
