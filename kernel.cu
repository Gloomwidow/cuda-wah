#include "hip/hip_runtime.h"
#include <cstdio>
#include <climits>
#include <thrust/remove.h>
#include "bit_functions.cuh"

struct zero
{
	__host__ __device__
		bool operator()(const int x)
	{
		return x == 0;
	}
};

__global__ void cuda_hello(){
    //printf("%d\n",blockIdx.x);
}


void CudaHello()
{
    printf("Hello extern!\n");
    cuda_hello<<<4,4>>>(); 
}


__global__ void ballot_warp_compress(UINT* input, UINT* output)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_id % 32;
    int warp_number = global_id / 32;

    bool is_zero = is_zeros(input[global_id]);
    bool is_one = is_ones(input[global_id]);

    UINT zeros = __ballot_sync(0xffffffff, is_zero);
    UINT ones = __ballot_sync(0xffffffff, is_one);
    zeros = reverse(zeros);
    ones = reverse(ones);
    bool checks_next = true;
    if (!is_zero && !is_one) //not suitable for compression
    {
        checks_next = false;
    }
    else
    {
        if (warp_id > 0)
        {
            int previous_zero = get_bit(zeros, warp_id-1);
            int previous_one = get_bit(ones, warp_id-1);
            if (previous_zero && is_zero) checks_next = false;
            else if (previous_one && is_one) checks_next = false;
        }
    }
    UINT add = 0;
    if (checks_next)
    {
        int pos = warp_id;
        while (pos <= 31)
        {
            int next_zero = get_bit(zeros, pos);
            int next_one = get_bit(ones, pos);
            if (is_zero && next_zero) add++;
            else if (is_one && next_one) add++;
            else break;
            pos++;
        }
        output[global_id] = get_compressed(add, is_one);
    }
    else
    {
        if (!is_zero && !is_one)output[global_id] = input[global_id]; //cant compress, writing literally
        else output[global_id] = 0; //'null' symbol after compression
    }
}


void BallotSyncWAH(UINT * input)
{
    int testSize = 64;
    UINT* test = new UINT[testSize];
    UINT* output = new UINT[testSize];
    for (int i = 0; i < testSize; i++)
    {
        int roll = rand() % 3;
        if (roll == 0)
        {
            test[i] = 0x7FFFFFFF; //all ones
            printf("1");
        }
        if (roll == 1)
        {
            test[i] = 256; // not valid for compression
            printf("x");
        }
        if (roll == 2)
        {
            test[i] = 0x00000000; // all zeros
            printf("0");
        }
    }
    printf("\n");
    UINT * d_test;
    UINT* d_output;
    hipMalloc((UINT**)&d_test, sizeof(UINT) * testSize);
    hipMalloc((UINT**)&d_output, sizeof(UINT) * testSize);
    hipMemcpy(d_test, test, sizeof(UINT)*testSize, hipMemcpyHostToDevice);
    ballot_warp_compress<<<testSize / 32, 32>>>(d_test,d_output);

    hipMemcpy(output, d_output, sizeof(UINT) * testSize, hipMemcpyDeviceToHost);

    UINT* end = thrust::remove_if(output, output + testSize, zero());

    for (int i = 0; i < end-output; i++)
    {
        printf("%u ", output[i]);
    }
    printf("\n");
    hipFree(d_test);
    hipFree(d_output);
    delete test;
    delete output;
}
