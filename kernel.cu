#include "hip/hip_runtime.h"
#include <cstdio>
#include <climits>
#include <thrust/remove.h>
#include "defines.h"
#include "methods.h"

//maximum amount of blocks we can compress in single block
#ifndef COMPRESS_MAX
#define COMPRESS_MAX 0x3FFFFFFF
#endif
//maximum input for logging
#ifndef LOGGING_MAX
#define LOGGING_MAX 128
#endif

struct zero
{
    __host__ __device__
        bool operator()(const int x)
    {
        return x == 0;
    }
};

//gets i-th bit in int
__host__ __device__ UINT get_bit(UINT src,int i)
{
    return (1 & (src >> (31-(i))));
}
//sets u-th bit to 1
__device__ UINT fill_bit(UINT src, int i)
{
    src |= 1UL << (31-i);
    return src;
}
//sets u-th bit to 0
__host__ __device__ UINT clear_bit(UINT src, int i)
{
    src &= ~(1UL << (31-i));
    return src;
}
//Compresses n blocks of 'bit'
//1st bit - is compressed flag
//2nd bit - what bit is in series (all 0 or all 1)
//3rd to end - block count
__device__ UINT get_compressed(UINT n, int bit)
{
    UINT rs = n;
    rs = fill_bit(rs, 0);
    if(bit) rs = fill_bit(rs, 1);
    return rs;
}

//returns amount of sequences compressed in src block
__host__ __device__ UINT compressed_count(UINT src)
{
    src = clear_bit(src,0);
    src = clear_bit(src,1);
    return src;
}

//checks if all bits in src are zero
__device__ bool is_zeros(UINT src)
{
    return src == 0;
}
//checks if all bits in src are ones
__device__ bool is_ones(UINT src)
{
    src = fill_bit(src, 0);
    return (~src) == 0;
}

__device__ UINT reverse(UINT src)
{
    UINT NO_OF_BITS = 32;
    UINT reverse_num = 0, i, temp;

    for (i = 0; i < NO_OF_BITS; i++)
    {
        temp = (src & (1 << i));
        if (temp) reverse_num |= (1 << ((NO_OF_BITS - 1) - i));
    }
    return reverse_num;
}



__global__ void ballot_warp_compress(UINT* input, UINT* output)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_id % 32;

    bool is_zero = is_zeros(input[global_id]);
    bool is_one = is_ones(input[global_id]);

    UINT zeros = __ballot_sync(0xffffffff, is_zero);
    UINT ones = __ballot_sync(0xffffffff, is_one);
    zeros = reverse(zeros);
    ones = reverse(ones);
    bool checks_next = true;
    if (!is_zero && !is_one) //not suitable for compression
    {
        checks_next = false;
    }
    else
    {
        if (warp_id > 0)
        {
            int previous_zero = get_bit(zeros, warp_id-1);
            int previous_one = get_bit(ones, warp_id-1);
            if (previous_zero && is_zero) checks_next = false;
            else if (previous_one && is_one) checks_next = false;
        }
    }
    UINT add = 0;
    if (checks_next)
    {
        int pos = warp_id;
        while (pos <= 31)
        {
            int next_zero = get_bit(zeros, pos);
            int next_one = get_bit(ones, pos);
            if (is_zero && next_zero) add++;
            else if (is_one && next_one) add++;
            else break;
            pos++;
        }
        output[global_id] = get_compressed(add, is_one);
    }
    else
    {
        if (!is_zero && !is_one)output[global_id] = input[global_id]; //cant compress, writing literally
        else output[global_id] = 0; //'null' symbol after compression
    }
}

__global__ void ballot_warp_compress(UINT* input, UINT* output)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = global_id % 32;

    bool is_zero = is_zeros(input[global_id]);
    bool is_one = is_ones(input[global_id]);

    UINT zeros = __ballot_sync(0xffffffff, is_zero);
    UINT ones = __ballot_sync(0xffffffff, is_one);
    zeros = reverse(zeros);
    ones = reverse(ones);
    bool checks_next = true;
    if (!is_zero && !is_one) //not suitable for compression
    {
        checks_next = false;
    }
    else
    {
        if (warp_id > 0)
        {
            int previous_zero = get_bit(zeros, warp_id-1);
            int previous_one = get_bit(ones, warp_id-1);
            if (previous_zero && is_zero) checks_next = false;
            else if (previous_one && is_one) checks_next = false;
        }
    }
    UINT add = 0;
    if (checks_next)
    {
        int pos = warp_id;
        while (pos <= 31)
        {
            int next_zero = get_bit(zeros, pos);
            int next_one = get_bit(ones, pos);
            if (is_zero && next_zero) add++;
            else if (is_one && next_one) add++;
            else break;
            pos++;
        }
        output[global_id] = get_compressed(add, is_one);
    }
    else
    {
        if (!is_zero && !is_one)output[global_id] = input[global_id]; //cant compress, writing literally
        else output[global_id] = 0; //'null' symbol after compression
    }
}

__global__ void ballot_warp_merge(int input_size, UINT* input, UINT* output)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id >= input_size) return;
    UINT curr = input[global_id];
    output[global_id] = 0;
    bool checks_next = true;
    if (!get_bit(curr, 0)) checks_next = false;
    else if (global_id > 0)
    {
        UINT prev = input[global_id - 1];
        if (get_bit(prev, 0) && (get_bit(prev, 1) == get_bit(curr, 1)))
        {
            checks_next = false;
        }
    }

    if (checks_next)
    {
        UINT bit = get_bit(curr, 1);
        int curr_output_pos = global_id;
        int pos = global_id + 1;
        UINT currAmount = compressed_count(curr);
        while (pos < input_size)
        {
            UINT iter = input[pos];
            if (get_bit(iter, 0) == 0) break;
            if (get_bit(iter, 1) != bit) break;
            UINT added = compressed_count(iter);
            currAmount += added; 
            pos++;
        }
        if (currAmount > 0)
        {
            output[curr_output_pos] = get_compressed(currAmount, bit);
        }
    }
    else if (!get_bit(curr, 0))
    {
        output[global_id] = curr;
    }
}


void BallotSyncWAH(UINT * input)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    int testSize = 32*(2);
    UINT* test = new UINT[testSize];
    UINT* output = new UINT[testSize];
    if (testSize <= LOGGING_MAX)
    {
        printf("Sequence for compression:\n");
        for (int i = 0; i < testSize; i++)
        {
            int roll = rand() % 1;
            if (roll == 0)
            {
                test[i] = 0x7FFFFFFF; //all ones
                printf("1");
            }
            if (roll == 2)
            {
                test[i] = 256; // not valid for compression
                printf("x");
            }
            if (roll == 1)
            {
                test[i] = 0x00000000; // all zeros
                printf("0");
            }
            if (((i + 1) % 32) == 0) printf("\n");
        }
        printf("\n");
    }
    UINT * d_test;
    UINT* d_output;
    hipMalloc((UINT**)&d_test, sizeof(UINT) * testSize);
    hipMalloc((UINT**)&d_output, sizeof(UINT) * testSize);

    hipMemcpy(d_test, test, sizeof(UINT)*testSize, hipMemcpyHostToDevice);
    ballot_warp_compress<<<testSize / 32, 32>>>(d_test,d_output);

    hipMemcpy(output, d_output, sizeof(UINT) * testSize, hipMemcpyDeviceToHost);
    UINT* end = thrust::remove_if(output, output + testSize, zero());
    int remove_count = end - output;
    if (testSize <= LOGGING_MAX)
    {
        printf("Sequence after warp-compression:\n");
        for (int i = 0; i < end - output; i++)
        {
            UINT c = compressed_count(output[i]);
            if (get_bit(output[i], 0)) printf("(%u,%u) ", c, get_bit(output[i], 1));
            else printf("x ");
        }
        printf("\n\n\n");
    }

    hipMemcpy(d_test, output, sizeof(UINT) * remove_count, hipMemcpyHostToDevice);
    ballot_warp_merge <<<(remove_count / 32)+1, 32 >> > (remove_count, d_test, d_output);
    hipMemcpy(output, d_output, sizeof(UINT) * testSize, hipMemcpyDeviceToHost);
    UINT* final_end = thrust::remove_if(output, output + remove_count, zero());
    if (testSize <= LOGGING_MAX)
    {
        printf("Sequence after global-compression:\n");
        for (int i = 0; i < final_end - output; i++)
        {
            UINT c = compressed_count(output[i]);
            if (get_bit(output[i], 0)) printf("(%u,%u) ", c, get_bit(output[i], 1));
            else printf("x ");
        }
        printf("\n");
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time passed: %fs\n", milliseconds /1000);
    hipFree(d_test);
    hipFree(d_output);
    delete test;
    delete output;
}


